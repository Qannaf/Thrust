﻿/*
#include "hip/hip_runtime.h"
#include ""



#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>


//vector

int main(void)
{
    // H has storage for 4 integers
    thrust::host_vector<int> H(4);

    // initialize individual elements
    H[0] = 14;
    H[1] = 20;
    H[2] = 38;
    H[3] = 46;

    // H.size() returns the size of vector H
    std::cout << "H has size " << H.size() << "\n";

    // print contents of H
    for (int i = 0; i < H.size(); i++)
        std::cout << "H[" << i << "] = " << H[i] << "\n";

    // resize H
    H.resize(2);

    std::cout << "H now has size " << H.size() << "\n";

    // Copy host_vector H to device_vector D
    thrust::device_vector<int> D = H;

    // elements of D can be modified
    D[0] = 99;
    D[1] = 88;

    // print contents of D
    for (int i = 0; i < D.size(); i++)
        std::cout << "D[" << i << "] = " << D[i] << "\n";

    // H and D are automatically deleted when the function returns
    return 0;
}
//*/

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include <iostream>


int main(void)
{
    int i(0);
    thrust::device_vector<int> D(10, 1);            // initialize all ten integers of a device_vector to 1
    thrust::fill(D.begin() + 3, D.begin() + 6, 0);  // set the first seven elements of a vector to 9

    thrust::host_vector<int> H(D.size());
    thrust::copy(D.begin(), D.end(), H.begin());    // copy all of H back to the beginning of D
    thrust::sequence(H.begin(), H.end());           // set the elements of H to 0, 1, 2, 3, ...


    // print D
    for (thrust::device_vector <int>::const_iterator it = D.begin(); it != D.end(); ++it)
    {
        std::cout << "D[" << i << "] = " << *it << "\t";
        i++;
    }
    for (int i = 0; i < H.size(); i++)
        std::cout << "H[" << i << "] = " << H[i] << "\t";

    return 0;
}
   
 